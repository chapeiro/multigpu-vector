#include <iostream>
#include <chrono>
#include <algorithm>
#include "common.cuh"
#include "select.cuh"

using namespace std;

#ifndef NVAL
#define NVAL (1024*64*1024*4)
#endif

constexpr int N = NVAL;

uint32_t unstable_select_gpu2(int32_t *src, int32_t *dst, uint32_t N, int32_t pred, const dim3 &dimGrid, const dim3 &dimBlock, hipEvent_t &start, hipEvent_t &stop){
    int32_t *buffer;
    int32_t grid_size = dimGrid.x * dimGrid.y * dimGrid.z;

    gpu(hipSetDevice(0));
    gpu(hipMalloc((void**)&buffer, (grid_size * 4 * WARPSIZE + 2)* sizeof(int32_t)));

    uint32_t* counters = (uint32_t *) (buffer + grid_size * 4 * WARPSIZE);
    
    // initialize global counters
    gpu(hipMemset(buffer + grid_size * 4 * WARPSIZE - grid_size, 0, (grid_size + 2) * sizeof(int32_t)));

    int32_t *buffer2;
    int32_t *dst2;

    gpu(hipSetDevice(1));
    hipMalloc((void**)&dst2, N/2*sizeof(int32_t));

    gpu(hipMalloc((void**)&buffer2, (grid_size * 4 * WARPSIZE + 2)* sizeof(int32_t)));

    uint32_t* counters2 = (uint32_t *) (buffer2 + grid_size * 4 * WARPSIZE);

    // initialize global counters
    gpu(hipMemset(buffer2 + grid_size * 4 * WARPSIZE - grid_size, 0, (grid_size + 2) * sizeof(int32_t)));

    gpu(hipSetDevice(0));
    hipEventRecord(start);

    size_t shared_mem = (9 * dimBlock.x * dimBlock.y + BRDCSTMEM(dimBlock) + ((dimBlock.x * dimBlock.y) / WARPSIZE))*sizeof(int32_t);

    // run kernel
    unstable_select<<<dimGrid, dimBlock, shared_mem>>>(src, dst, N/2, pred, buffer, counters, counters+1);

    gpu(hipSetDevice(1));

    // run kernel
    hipStreamWaitEvent(NULL, start, 0); //only for correctly counting the time for both kernels
    unstable_select<<<dimGrid, dimBlock, shared_mem>>>(src+N/2, dst2, N/2, pred, buffer2, counters2, counters2+1);
    
    gpu(hipSetDevice(0));
#ifndef NDEBUG
    gpu(hipPeekAtLastError()  );
    gpu(hipDeviceSynchronize());
#endif

    // wait to read counters from device
    uint32_t h_counters[2];
    gpu(hipMemcpy(h_counters, counters, 2 * sizeof(uint32_t), hipMemcpyDefault));
    uint32_t h_output_size = h_counters[0];
    uint32_t h_buffer_end  = h_counters[1];
    uint32_t h_buffer_start= (h_counters[1]/(4*WARPSIZE))*(4*WARPSIZE);
    uint32_t h_buffer_size = h_buffer_end - h_buffer_start;
    assert(h_buffer_start % (4*WARPSIZE) == 0);
    assert(h_buffer_end >= h_buffer_start);
    assert(h_buffer_size < 4*WARPSIZE);

    // combine results
    if (h_buffer_size > 0) hipMemcpy(dst+h_output_size, buffer+h_buffer_start, h_buffer_size * sizeof(int32_t), hipMemcpyDefault);

    gpu(hipSetDevice(1));
#ifndef NDEBUG
    gpu(hipPeekAtLastError()  );
    gpu(hipDeviceSynchronize());
#endif

    gpu(hipMemcpy(h_counters, counters2, 2 * sizeof(uint32_t), hipMemcpyDefault));
    uint32_t h_output_size2 = h_counters[0];
    uint32_t h_buffer_end2  = h_counters[1];
    uint32_t h_buffer_start2= (h_counters[1]/(4*WARPSIZE))*(4*WARPSIZE);
    uint32_t h_buffer_size2 = h_buffer_end2 - h_buffer_start2;
    assert(h_buffer_start2 % (4*WARPSIZE) == 0);
    assert(h_buffer_end2 >= h_buffer_start2);
    assert(h_buffer_size2 < 4*WARPSIZE);

    // combine results
    if (h_buffer_size2 > 0) hipMemcpy(dst+h_output_size+h_buffer_size+h_output_size2, buffer2+h_buffer_start2, h_buffer_size2 * sizeof(int32_t), hipMemcpyDefault);
    

    gpu(hipSetDevice(1));
    gpu(hipMemcpy(dst+h_output_size+h_buffer_size, dst2, h_output_size2*sizeof(int32_t), hipMemcpyDefault));
    
    gpu(hipSetDevice(0));
    hipEventRecord(stop);
    
    gpu(hipSetDevice(0));
    gpu(hipFree(dst2));
    return h_output_size+h_buffer_size+h_output_size2+h_buffer_size2;
}


int32_t *a;
int32_t *b;

int main(){
    gpu(hipSetDevice(1));
    gpu(hipFree(0)); //initialize devices on demand
    gpu(hipSetDevice(0));
    gpu(hipFree(0)); //initialize devices on demand
    srand(time(0));

    a = (int32_t*) malloc(N*sizeof(int32_t));
    b = (int32_t*) malloc(N*sizeof(int32_t));

    for (int i = 0 ; i < N ; ++i) a[i] = rand() % 100 + 1;
    
    // char *ad;
    // int *bd;
    // const int csize = N*sizeof(char);
    // const int isize = N*sizeof(int);

    double millis = 0;
    {
        auto start = chrono::high_resolution_clock::now();
        stable_select_cpu(a, b, N);
        auto end   = chrono::high_resolution_clock::now();
        auto diff = end - start;

        millis = chrono::duration<double, milli>(diff).count();
        cout << millis << " ms" << endl;
    }
    int32_t *a_pinned;
    int32_t *b_pinned;
    hipEvent_t start, stop, start1, stop1, start2, stop2;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    dim3 dimBlock(1024, 1 );
    dim3 dimGrid( 8, 1 );

    hipHostMalloc((void**)&a_pinned, N*sizeof(int32_t));
    hipHostMalloc((void**)&b_pinned, N*sizeof(int32_t));

    memcpy(a_pinned, a, N*sizeof(int32_t));

#ifndef NTESTUVA
    int results1;
    {
    auto ts = chrono::high_resolution_clock::now();
     results1 = unstable_select_gpu2(a_pinned, b_pinned, N, 50, dimGrid, dimBlock, start1, stop1);
    auto te   = chrono::high_resolution_clock::now();
    auto diff = te - ts;
    auto millis = chrono::duration<double, milli>(diff).count();
    cout << millis << " ms" << endl;
    }
#else
    int results1 = 0;
#endif

#ifndef NTESTMEMCPY
    int32_t *a_dev, *b_dev;

    gpu(hipMalloc( (void**)&a_dev, N*sizeof(int32_t)));
    gpu(hipMalloc( (void**)&b_dev, N*sizeof(int32_t)));

    hipEventRecord(start);
    gpu(hipMemcpy( a_dev, a_pinned, N*sizeof(int32_t), hipMemcpyDefault));
    
    int results2 = unstable_select_gpu(a_dev, b_dev, N, 50, dimGrid, dimBlock, start2, stop2);

    gpu(hipMemcpy(a_pinned, b_dev, N*sizeof(int32_t), hipMemcpyDefault));
    hipEventRecord(stop);

    gpu(hipFree(a_dev));
    gpu(hipFree(b_dev));
#else
    int results2 = 0;
#endif

    hipEventSynchronize(stop);

    hipDeviceSynchronize();
#ifndef NDEBUG
    int results = N;
    for (int i = 0 ; i < N ; ++i) {
        if (b[i] == -1) {
            results = i;
            break;
        } else {
            assert(b[i] <= 50);
            assert(b[i] > 0);
        }
    }
#ifndef NTESTUVA
    for (int i = 0 ; i < results1 ; ++i) {
        if (b_pinned[i] <= 0 || b_pinned[i] > 50){
            cout << b_pinned[i] << " " << i << endl;
        }
        // assert(b_pinned[i] <= 50);
        // assert(b_pinned[i] > 0);
    }
#endif
#ifndef NTESTMEMCPY
    for (int i = 0 ; i < results2 ; ++i) {
        if (a_pinned[i] <= 0 || a_pinned[i] > 50){
            cout << a_pinned[i] << " " << i << endl;
        }
        // assert(b_pinned[i] <= 50);
        // assert(b_pinned[i] > 0);
    }
#endif
    cout << results << " " << results1 << " " << results2 << " " << a_pinned[4] << endl;

    // assert(results1 == results2);
    if (results != results1){
        cout << "Wrong results!!!!!!" << endl;
    } else {
        sort(b_pinned, b_pinned + results);
        sort(b       , b        + results);
        for (int i = 0 ; i < results ; ++i){
            if (b[i] != b_pinned[i]){
                cout << "Wrong result: " << b_pinned[i] << " (vs " << b[i] << ") @" << i << " !!!!!!" << endl;
                exit(-1);
            }
        }
    }
#endif
    gpu(hipHostFree(a_pinned));
    gpu(hipHostFree(b_pinned));

    float milliseconds1 = 0;
    hipEventElapsedTime(&milliseconds1, start, stop);
    cout << milliseconds1 << endl;
    float milliseconds2 = 0;
    hipEventElapsedTime(&milliseconds2, start1, stop1);
    cout << milliseconds2 << endl;
    float milliseconds3 = 0;
    hipEventElapsedTime(&milliseconds3, start2, stop2);
    cout << milliseconds3 << endl;

    cout << endl;
    cout << millis/milliseconds1 << endl;
    cout << millis/milliseconds2 << endl;
    cout << millis/milliseconds3 << endl;
    
    hipDeviceSynchronize();
    hipDeviceReset();

    return EXIT_SUCCESS;
}
