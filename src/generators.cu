#include "hip/hip_runtime.h"
#include "generators.cuh"
#include "buffer_manager.cuh"

#include <iostream>
#include <chrono>

using namespace std;

__host__ generator::generator(Operator * parent, int32_t *src, uint32_t N):
        parent(parent), src(src), N(N){
    // parent->open();

    // hipStream_t strm;
    gpu(hipStreamCreateWithFlags(&strm, hipStreamNonBlocking));
    
    // buffer_pool<int32_t>::buffer_t ** buff_ret;
    gpu(hipHostMalloc(&buff_ret, sizeof(buffer_pool<int32_t>::buffer_t *)));

    // buffer_pool<int32_t>::buffer_t::inspector_t insp(strm);
    // while (N > 0){
    //     buffer_pool<int32_t>::buffer_t * buff = buffer_manager<int32_t>::get_buffer();

    //     int m = min(N, buff_size);

    //     insp.load(buff, true);
    //     insp.overwrite(src, m);

    //     insp.save(buff, true);

    //     parent->consume(buff);
    //     // variant::apply_visitor(push(buff), *parent);
    //     // push()(*parent); //->consume(buff);

    //     N   -= m;
    //     src += m;
    // }
    // gpu(hipStreamSynchronize(strm));
    // gpu(hipStreamDestroy(strm));
    // gpu(hipHostFree(buff_ret));

    // parent->close();
}

__host__ __device__ void generator::consume(buffer_pool<int32_t>::buffer_t * data){
#ifdef __CUDA_ARCH__
    assert(false);
#else
    buffer_pool<int32_t>::buffer_t::inspector_t insp(strm);
    auto start = chrono::system_clock::now();
    while (N > 0){
        buffer_pool<int32_t>::buffer_t * buff = buffer_manager<int32_t>::get_buffer();

        int m = min(N, buffer_pool<int32_t>::buffer_t::capacity());

        insp.load(buff, true);
        insp.overwrite(src, m);

        insp.save(buff, true);

        parent->consume(buff);

        N   -= m;
        src += m;
    }
    auto end   = chrono::system_clock::now();
    cout << chrono::duration_cast<chrono::milliseconds>(end - start).count() << "ms" << endl;
#endif
}

__host__ __device__ void generator::join(){
#ifdef __CUDA_ARCH__
    assert(false);
#else
    auto start = chrono::system_clock::now();
    gpu(hipStreamSynchronize(strm));
    parent->close();
    auto end   = chrono::system_clock::now();
    cout << chrono::duration_cast<chrono::milliseconds>(end - start).count() << "ms" << endl;
#endif
}

__host__ generator::~generator(){
    gpu(hipStreamDestroy(strm));
    gpu(hipHostFree(buff_ret));
}



// template<>
// __host__ __device__ void push::operator()<generator *>(generator * op) const{
//    op->consume(NULL);
// }
