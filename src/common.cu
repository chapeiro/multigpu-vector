#include "common.cuh"

__host__ int get_device(const void *p){
    hipPointerAttribute_t attrs;
    hipError_t error = hipPointerGetAttributes(&attrs, p);
    if (error == hipErrorInvalidValue) return -1;
    gpu(error);
    return (attrs.memoryType == hipMemoryTypeHost) ? -1 : attrs.device;
}