#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <algorithm>
#include <vector>
#include <thread>
#include <mutex>
#include <condition_variable>
#include <chrono>
#include "common.cuh"
#include "select.cuh"
#include <iomanip>
#include "buffer_manager.cuh"
#include <hip/hip_runtime_api.h>
#include <chrono>
// #include <functional>

using namespace std;

#ifndef NVAL
#define NVAL (64*1024*1024*4)
#endif

constexpr int N = NVAL;



vector<pair<int32_t *, uint32_t>> data_pool;

mutex data_pool_mutex;

condition_variable cv;
unsigned int remaining_sources = 0;

vector<buffer_pool<int32_t>::buffer_t *> data_pool2;

mutex data_pool_mutex2;

condition_variable cv2;
unsigned int remaining_sources2 = 0;

mutex data_pool_mutex_cons_ready;

condition_variable cv_cons_ready;
unsigned int consumers_not_loaded = 2;//4;

__host__ void consume(unstable_select_gpu<> *filter, buffer_pool<int32_t> *outpool){
    set_device_on_scope(filter->device);
    outpool->register_producer(filter);
    {
        unique_lock<mutex> lock(data_pool_mutex_cons_ready);
        --consumers_not_loaded;
        lock.unlock();
        cv_cons_ready.notify_all();
    }
    auto start = std::chrono::system_clock::now();
    do {
        unique_lock<mutex> lock(data_pool_mutex);

        cout << data_pool.size() << endl;
        cv.wait(lock, []{return !data_pool.empty() || (data_pool.empty() && remaining_sources == 0);});

        if (data_pool.empty()){
            assert(remaining_sources == 0);
            lock.unlock();
            break;
        }

        pair<int32_t *, uint32_t> p = data_pool.back();
        data_pool.pop_back();
        lock.unlock();
        cout << "starting filter instance..." << endl;
        filter->next(p.first, p.second);
        hipStreamSynchronize(filter->stream);
        cout << "ended filter instance..." << endl;
        // outpool->acquire_buffer_blocked();
        // filter->next(dst, p.first, p.second);
    } while (true);
    filter->next();
    hipStreamSynchronize(filter->stream);
    outpool->unregister_producer(filter);
    auto end = std::chrono::system_clock::now();
    auto dur = end - start;
    cout << "duration: " << dur.count() << endl;
}


__host__ void generator2(buffer_pool<int32_t> *src, int device = 0){
    {
        unique_lock<mutex> lock(data_pool_mutex_cons_ready);

        cv_cons_ready.wait(lock, []{return consumers_not_loaded == 0;});

        lock.unlock();
    }
    // --remaining_sources2;return;
    set_device_on_scope d(device);
    buffer_pool<int32_t>::buffer_t ** buff_ret;
    hipHostMalloc(&buff_ret, sizeof(buffer_pool<int32_t>::buffer_t *));
    hipStream_t strm;
    hipStreamCreateWithFlags(&strm, hipStreamNonBlocking);

    do {
        cout << "poll " << endl;
        buffer_pool<int32_t>::buffer_t * buff = src->h_acquire_buffer_blocked(buff_ret, strm);
        cout << "]]]]]]" << buff << endl;
        if (buff == (buffer_pool<int32_t>::buffer_t *) 1) {
            // this_thread::sleep_for(chrono::microseconds(100));
            continue;
        }
        if (!src->is_valid(buff)) break;

        unique_lock<mutex> lock(data_pool_mutex2);
        data_pool2.emplace_back(buff);
        cv2.notify_all();
        lock.unlock();
    } while(true);
    --remaining_sources2;
    cv2.notify_all();

    hipStreamDestroy(strm);
    hipHostFree(buff_ret);
}


__host__ void consume2(int32_t *dst, uint32_t *res, int device = 0){
    set_device_on_scope d(device);
    hipStream_t strm2;
    hipStreamCreateWithFlags(&strm2, hipStreamNonBlocking);
    // buffpool->register_producer(NULL);
    buffer_pool<int32_t>::buffer_t::inspector_t insp(strm2);
    do {
        unique_lock<mutex> lock(data_pool_mutex2);

        cv2.wait(lock, []{return !data_pool2.empty() || (data_pool2.empty() && remaining_sources2 == 0);});

        if (data_pool2.empty()){
            assert(remaining_sources2 == 0);
            lock.unlock();
            break;
        }

        buffer_pool<int32_t>::buffer_t *p = data_pool2.back(); //FIXME: release buffer back to device
        data_pool2.pop_back();

        insp.load(p, true);

        uint32_t cnt = insp.count();

        const int32_t  *data = (const int32_t *) insp.data();
        uint32_t start = *res;
        *res += cnt;
        lock.unlock();

        cout << data << endl;
        gpu(hipMemcpyAsync(dst+start, data, sizeof(int32_t)*cnt, hipMemcpyDefault, strm2));
        cout << "---------------------------------------------------------------consumed: " << dec << (uint64_t) *res << endl;

        buffer_manager<int32_t>::release_buffer(p, strm2);
        // buffpool->h_release_buffer(p, strm2);
    } while (true);
    gpu(hipStreamSynchronize(strm2));
    // buffpool->unregister_producer(NULL);
    hipStreamDestroy(strm2);
}

void generate_data(int32_t *src, uint32_t N, uint32_t buff_size){
    for (uint32_t i = 0 ; i < N ; i += buff_size){
        unique_lock<mutex> lock(data_pool_mutex);
        data_pool.emplace_back(src+i, min(N - i, buff_size));
        lock.unlock();
        cv.notify_all();
        // this_thread::sleep_for(chrono::milliseconds(250));
    }
    --remaining_sources;
    cv.notify_all();
}

__host__ uint32_t unstable_select_gpu_caller2(int32_t *src, int32_t *dst, uint32_t N, int32_t pred, const dim3 &dimGrid, const dim3 &dimBlock, chrono::microseconds &dur){
    data_pool.clear();

    set_device_on_scope d(1);

    remaining_sources = 1;
    remaining_sources2 = 2;

    hipStream_t sta, stb;//, stc, std;
    {
        set_device_on_scope d(0);
        hipStreamCreateWithFlags(&sta, hipStreamNonBlocking);
    }
    {
        set_device_on_scope d(1);
        hipStreamCreateWithFlags(&stb, hipStreamNonBlocking);
    }
    // hipStreamCreateWithFlags(&stc, hipStreamNonBlocking);
    // hipStreamCreateWithFlags(&std, hipStreamNonBlocking);

    uint32_t res1 = 0;

    thread g(generate_data, src, N, DEFAULT_BUFF_CAP);//WARPSIZE*64);

    buffer_pool<int32_t> *outpool0 = cuda_new<buffer_pool<int32_t>>(0, 1024, 0, 0);
    buffer_pool<int32_t> *outpool1 = cuda_new<buffer_pool<int32_t>>(1, 1024, 0, 1);

    unstable_select_gpu<> filter1(dimGrid, dimBlock, outpool0, sta, 0);
    unstable_select_gpu<> filter2(dimGrid, dimBlock, outpool1, stb, 1);
    // unstable_select_gpu<> filter3(dimGrid, dimBlock, outpool, stc);
    // unstable_select_gpu<> filter4(dimGrid, dimBlock, outpool, stc);

    gpu(hipDeviceSynchronize());

    hipProfilerStart();
    // hipEventRecord(start, 0);

    chrono::system_clock::time_point start = chrono::system_clock::now();

    thread t1(consume, &filter1, outpool0);
    thread t2(consume, &filter2, outpool1);
    // thread t2b(consume, &filter3, outpool);
    // thread t2c(consume, &filter4, outpool);
    thread t30(generator2, outpool0, 0);
    thread t31(generator2, outpool1, 1);
    thread t4(consume2, dst, &res1, 1);
    t1.join();
    t2.join();
    // t2b.join();
    // t2c.join();
    t30.join();
    t31.join();
    t4.join();
    g.join();

    chrono::system_clock::time_point end = chrono::system_clock::now();
    dur = chrono::duration_cast<chrono::microseconds>(end - start);
    // hipEventRecord(stop, 0);
    hipProfilerStop();

    cuda_delete(outpool0);
    cuda_delete(outpool1);
    
    return res1;
}

int32_t *a;
int32_t *b;

int main(){
    setbuf(stdout, NULL);
    // gpu(hipSetDeviceFlags(hipDeviceScheduleYield));
    srand(time(0));
    buffer_manager<int32_t>::init();

    cout << "asdasf " << endl;
    
    a = (int32_t*) malloc(N*sizeof(int32_t));
    b = (int32_t*) malloc(N*sizeof(int32_t));

    for (int i = 0 ; i < N ; ++i) a[i] = rand() % 100 + 1;
    
    // char *ad;
    // int *bd;
    // const int csize = N*sizeof(char);
    // const int isize = N*sizeof(int);

    double millis = 0;
#ifndef NCPU
    {
        auto start = chrono::high_resolution_clock::now();
        stable_select_cpu(a, b, N);
        auto end   = chrono::high_resolution_clock::now();
        auto diff = end - start;

        millis = chrono::duration<double, milli>(diff).count();
        cout << millis << " ms " << endl;
    }
#ifndef NCPU
    int results = N;
    for (int i = 0 ; i < N ; ++i) {
        if (b[i] == -1) {
            results = i;
            break;
        } else {
            assert(b[i] <= 50);
            assert(b[i] > 0);
        }
    }
    cout << results << endl;
#endif
#endif
    int32_t *a_pinned;
    int32_t *b_pinned;
    hipEvent_t start, stop, start1, stop1, start2, stop2;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    {
        set_device_on_scope d(1);
        hipEventCreate(&start1);
        hipEventCreate(&stop1);
    }
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    dim3 dimBlock(1024, 1 );
    dim3 dimGrid( 8, 1 );

    hipHostMalloc((void**)&a_pinned, N*sizeof(int32_t));
    hipHostMalloc((void**)&b_pinned, N*sizeof(int32_t));

    memcpy(a_pinned, a, N*sizeof(int32_t));

#ifndef NTESTUVA
    chrono::microseconds dur;
    int results1 = unstable_select_gpu_caller2(a_pinned, b_pinned, N, 50, dimGrid, dimBlock, dur);
#else
    int results1 = 0;
#endif

#ifndef NTESTMEMCPY
    int32_t *a_dev, *b_dev;

    gpu(hipMalloc( (void**)&a_dev, N*sizeof(int32_t)));
    gpu(hipMalloc( (void**)&b_dev, N*sizeof(int32_t)));

    hipEventRecord(start);
    gpu(hipMemcpy( a_dev, a_pinned, N*sizeof(int32_t), hipMemcpyDefault));
    
    int results2 = unstable_select_gpu_caller2(a_dev, b_dev, N, 50, dimGrid, dimBlock, start2, stop2);

    gpu(hipMemcpy(a_pinned, b_dev, N*sizeof(int32_t), hipMemcpyDefault));
    hipEventRecord(stop);

    gpu(hipFree(a_dev));
    gpu(hipFree(b_dev));
#else
    int results2 = 0;
#endif

    hipEventSynchronize(stop);

    hipDeviceSynchronize();
#ifndef NDEBUG
    // int results = N;
    // for (int i = 0 ; i < N ; ++i) {
    //     if (b[i] == -1) {
    //         results = i;
    //         break;
    //     } else {
    //         assert(b[i] <= 50);
    //         assert(b[i] > 0);
    //     }
    // }
#ifndef NTESTUVA
    for (int i = 0 ; i < results1 ; ++i) {
        if (b_pinned[i] <= 0 || b_pinned[i] > 50){
            cout << b_pinned[i] << " " << i << endl;
        }
        // assert(b_pinned[i] <= 50);
        // assert(b_pinned[i] > 0);
    }
#endif
#ifndef NTESTMEMCPY
    for (int i = 0 ; i < results2 ; ++i) {
        if (a_pinned[i] <= 0 || a_pinned[i] > 50){
            cout << a_pinned[i] << " " << i << endl;
        }
        // assert(b_pinned[i] <= 50);
        // assert(b_pinned[i] > 0);
    }
#endif
#endif
    cout << results << " " << results1 << " " << results2 << " " << a_pinned[4] << endl;

    float milliseconds1 = 0;
    hipEventElapsedTime(&milliseconds1, start, stop);
    cout << milliseconds1 << endl;
    float milliseconds2 = dur.count()/1000.0;
    // hipEventElapsedTime(&milliseconds2, start1, stop1);
    cout << milliseconds2 << endl;
    float milliseconds3 = 0;
    hipEventElapsedTime(&milliseconds3, start2, stop2);
    cout << milliseconds3 << endl;

    cout << endl;
    cout << millis/milliseconds1 << endl;
    cout << millis/milliseconds2 << endl;
    cout << millis/milliseconds3 << endl;
#ifndef NDEBUG
    // assert(results1 == results2);
    if (results != results1){
        cout << "Wrong results!!!!!!" << endl;
    } else {
        cout << "Skipping checking results..." << endl;
        return 0;
#ifndef __CUDA_ARCH__
        sort(b_pinned, b_pinned + results1);
        sort(b       , b        + results);
        int errors = 0;
        int i1 = 0;
        int i2 = 0;
        while (i1 < results && i2 < results1){
            if (b[i1] != b_pinned[i2]){
                ++errors;
                if (b[i1] < b_pinned[i2]){
                    cout << "Lost  " << b_pinned[i2] << " (vs " << b[i1] << ") @" << i2 << " !!!!!!" << endl;
                    ++i1;
                } else {
                    cout << "Extra " << b_pinned[i2] << " (vs " << b[i1] << ") @" << i2 << " !!!!!!" << endl;
                    ++i2;
                }
            } else {
                ++i1;
                ++i2;
            }
        }
        int missing = (results - i1) + (results - i2);
        errors += missing;
        if (missing) cout << "Missing : " << missing << endl;
        if (errors ) {
            cout << "Total errors: " << errors << endl;
            exit(-1);
        }
#endif
    }
#endif
    gpu(hipHostFree(a_pinned));
    gpu(hipHostFree(b_pinned));
    
    hipDeviceSynchronize();
    hipDeviceReset();

    return EXIT_SUCCESS;
}
