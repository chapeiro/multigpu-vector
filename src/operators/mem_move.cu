#include "mem_move.cuh"
#include "../buffer_manager.cuh"
#include "../numa_utils.cuh"


template<typename T>
void make_mem_move(const T * __restrict__ &src, int target_device, cnt_t N, hipStream_t strm){
    int dev = get_device(src);

    if (dev != target_device){
        set_device_on_scope d(dev);

        if (dev >= 0) set_affinity_local_to_gpu(dev);

        T * buff = buffer_manager<int32_t>::h_get_buffer(target_device);

        buffer_manager<int32_t>::overwrite     (buff, src, N, strm, true);
        // buffer_manager<int32_t>::release_buffer(src                     );

        src = buff;
    }
}

template<typename T>
__host__ mem_move<T>::mem_move(h_operator<T> parent, int target_device): parent(parent), target_device(target_device), t(new thread([]{})){
    gpu(hipStreamCreateWithFlags(&strm, hipStreamNonBlocking));
}

template<typename T>
__host__ void mem_move<T>::consume(const T * __restrict__ src, cnt_t N, vid_t vid, cid_t cid){
    make_mem_move(src, target_device, N, strm);

    t->join();
    delete t;

    t = new thread([src, N, vid, cid, this](){ //what about src ? if no memcpy ? who will free it ?
        parent.consume(src, N, vid, cid);
    });
}

template<typename T>
template<typename Tw>
__host__ void mem_move<T>::serialize_with(mem_move<Tw> *t){
    gpu(hipStreamSynchronize(strm));
    gpu(hipStreamDestroy(strm));
    strm  = t->strm;
}

template<typename T>
__host__ void mem_move<T>::close(){
    t->join();
    delete t;
    parent.close();
}

template<typename T>
__host__ void mem_move<T>::open(){
    parent.open();
}

template class mem_move<int32_t>;
template __host__ void mem_move<int32_t>::serialize_with(mem_move<int32_t> *);
template __host__ void mem_move<int32_t>::serialize_with(mem_move<sel_t  > *);



template<int index, typename... Ts>
struct mem_g_move_tuple {
    void operator() (tuple<Ts...>& t, int target_device, cnt_t N, hipStream_t strm) {
        make_mem_move(get<index>(t), target_device, N, strm);

        mem_g_move_tuple<index - 1, Ts...>{}(t, target_device, N, strm);
    }
};

template<typename... Ts>
struct mem_g_move_tuple<-1, Ts...> {
    void operator() (tuple<Ts...>& t, int target_device, cnt_t N, hipStream_t strm) {}
};

template<typename... Ts>
void mem_g_move(tuple<Ts...>& t, int target_device, cnt_t N, hipStream_t strm) {
    const auto size = tuple_size<tuple<Ts...>>::value;
    mem_g_move_tuple<size - 1, Ts...>{}(t, target_device, N, strm);
}


template<typename... T>
__host__ mem_multimove<T...>::mem_multimove(h_operator<T...> parent, int target_device): parent(parent), target_device(target_device), t(new thread([]{})){
    gpu(hipStreamCreateWithFlags(&strm, hipStreamNonBlocking));
}



template<typename... T>
__host__ void mem_multimove<T...>::consume(const T * __restrict__ ... src, cnt_t N, vid_t vid, cid_t cid){
    std::tuple<const T *...> tsrc(make_tuple(src...));
    mem_g_move(tsrc, target_device, N, strm);

    hipCtx_t ctx;
    gpu(hipCtxGetCurrent(&ctx));

    t->join();
    delete t;

    t = new thread([tsrc, N, vid, cid, ctx, this](){ //what about src ? if no memcpy ? who will free it ?
        gpu(hipCtxSetCurrent(ctx));
        parent.consume(tsrc, N, vid, cid);
    });
}

template<typename... T>
template<typename... Tw>
__host__ void mem_multimove<T...>::serialize_with(mem_multimove<Tw...> *t){
    gpu(hipStreamSynchronize(strm));
    gpu(hipStreamDestroy(strm));
    strm  = t->strm;
}

template<typename... T>
__host__ void mem_multimove<T...>::close(){
    t->join();
    delete t;
    parent.close();
}

template<typename... T>
__host__ void mem_multimove<T...>::open(){
    parent.open();
}


template class mem_multimove<int32_t, int32_t>;
template __host__ void mem_multimove<int32_t, int32_t>::serialize_with(mem_multimove<int32_t, int32_t> *);

template class mem_multimove<int32_t, int32_t, int32_t, int32_t>;
template __host__ void mem_multimove<int32_t, int32_t, int32_t, int32_t>::serialize_with(mem_multimove<int32_t, int32_t, int32_t, int32_t> *);