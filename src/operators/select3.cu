#include "hip/hip_runtime.h"
#include "select3.cuh"

using namespace std;

template<size_t warp_size, typename F, typename... T>
__host__ unstable_select<warp_size, F, T...>::unstable_select(d_operator_t * parent, F f, int grid_size, int dev): 
        parent(parent), buffer_size(0), finished(0), filt(f){
    // output = cuda_new<output_composer<warp_size, T>>(dev, parent, dev);

    assert(dev >= 0);
    set_device_on_scope d(dev);

    gpu(hipMalloc((void**)&buffer, ((grid_size + 1) * 4 * warp_size + 3) * sizeof(int32_t)));//FIXME: corrent type

    gpu(hipMemset(buffer, 0, ((grid_size + 1) * 4 * warp_size + 3) * sizeof(int32_t)));//FIXME: corrent type
    // gpu(hipMemset(buffer + (grid_size + 1) * (4 * warp_size - 1), 0, (grid_size + 4) * sizeof(int32_t)));//FIXME: corrent type
}

template<size_t warp_size, typename F, typename... T>
__device__ void unstable_select<warp_size, F, T...>::consume_warp(const T *... src, unsigned int N){
    extern __shared__ int32_t s[];

    const int32_t warpid            = get_warpid();
    const int32_t laneid            = get_laneid();
    // const int32_t width             = blockDim.x * blockDim.y;

    const int32_t prevwrapmask      = (1 << laneid) - 1;

    // volatile int32_t *fcount      = (int32_t *) s;
    // volatile int32_t *wrapoutbase = (int32_t *) (s + (width + warpSize - 1) / warpSize);
    volatile int32_t *wrapoutbase   = ((int32_t *) s);
    // volatile int32_t *fcount        = ((int32_t *) s) + 5 * warp_size * (((width + warp_size - 1) / warp_size) + 1);
    volatile int32_t *wrapoutput    = wrapoutbase + 5 * warp_size * warpid;

    int32_t filterout = wrapoutput[5*warp_size - 1];

    F filter(filt);

    vec4 x;
    #pragma unroll
    for (int k = 0 ; k < 4 ; ++k){
        if (k*warpSize + laneid < N){
            //compute predicate
            x.i[k] = get<0>(make_tuple(src[k*warpSize + laneid]...));
        }
    }

    #pragma unroll
    for (int k = 0 ; k < 4 ; ++k){
        bool predicate = false;
        if (k*warpSize + laneid < N){
            //compute predicate
            predicate = filter(x.i[k]);
        }

        //aggregate predicate results
        int32_t filter = __ballot(predicate); //filter now contains set bits only for threads

        int32_t newpop = __popc(filter);

        assert(filterout < 4*warpSize);

        //compute position of result
        if (predicate){
            int32_t offset = filterout + __popc(filter & prevwrapmask);
            assert(offset >= 0);
            assert(offset <  5*warpSize);
            wrapoutput[offset] = x.i[k];// x.i[k];//input[blockDim.x*k + i];
        }

        filterout += newpop;

        if (filterout >= 4*warpSize){
            // output.push(wrapoutput);
            parent->consume_warp((const int32_t *) wrapoutput, 4*warpSize);

            wrapoutput[laneid]             = wrapoutput[laneid + 4*warpSize];
            filterout                     -= 4*warpSize;
        }

        // __syncthreads(); //FIXME: this should not be needed, but racecheck produces errors without it
    }

    if (laneid == 0) wrapoutput[5*warp_size - 1] = filterout;
}

template<size_t warp_size, typename F, typename... T>
__device__ void unstable_select<warp_size, F, T...>::consume_close(){
    extern __shared__ int32_t s[];

    const int32_t warpid            = get_warpid();
    const int32_t laneid            = get_laneid();
    // const int32_t width             = blockDim.x * blockDim.y;

    const int32_t gridwidth         = gridDim.x     * gridDim.y ;

    // volatile int32_t *fcount      = (int32_t *) s;
    // volatile int32_t *wrapoutbase = (int32_t *) (s + (width + warpSize - 1) / warpSize);
    volatile int32_t *wrapoutbase = ((int32_t *) s);
    volatile int32_t *wrapoutput  = wrapoutbase + 5 * warp_size * warpid;
    volatile int32_t *fcount      = wrapoutput  + 5 * warp_size - 1;

    int32_t filterout = *fcount;

    for (int32_t m = 1 ; m <= 5 ; ++m){ //fixme: not until 5, but until ceil(log(max warpid)) ? also ceil on target_filter_out condition
        int32_t mask = (1 << m) - 1;

        if (!(warpid & mask)){
            int32_t target_wrapid               = warpid + (1 << (m - 1));
            
            volatile int32_t *target_wrapoutput = wrapoutbase + 5 * warpSize * target_wrapid;

            int32_t target_filter_out           = (target_wrapid < blockDim.x * blockDim.y/warpSize) ? target_wrapoutput[5*warp_size - 1] : 0;
            int32_t target_filter_out_rem       = target_filter_out;

            assert(target_filter_out < 4*warpSize);
            assert(target_filter_out >= 0);

            for (int32_t k = 0; k < (target_filter_out + warpSize - 1)/warpSize ; ++k){
                assert(k < 4);
                if (laneid + k * warpSize < target_filter_out) {
                    assert(filterout + laneid < 5*warpSize);
                    wrapoutput[filterout + laneid] = target_wrapoutput[laneid + k * warpSize];
                }
                int32_t delta = min(target_filter_out_rem, warpSize);
                target_filter_out_rem -= delta;
                filterout += delta;

                if (filterout >= 4*warpSize){
                    // output.push(wrapoutput);
                    parent->consume_warp((const int32_t *) wrapoutput, 4*warpSize);

                    wrapoutput[laneid]             = wrapoutput[laneid + 4*warpSize];
                    filterout                     -= 4*warpSize;
                }
            }

            //no __syncthreads is needed here due to the pattern of accesses on fcount
            if (laneid == 0) *fcount = filterout;
        }
        __syncthreads();
    }

    if (warpid == 0 && filterout){
        int32_t elems_old;
        if (laneid == 0) {
            elems_old = atomicAdd((uint32_t *) &buffer_size, filterout);
            assert((elems_old + filterout) <= (gridwidth + 1) * (4 * warp_size - 1));
        }
        elems_old = brdcst(elems_old, 0);

        volatile int32_t * buffoff = buffer  + elems_old;
        volatile int32_t * aligned = (int32_t *) round_up((uintptr_t) buffoff, warpSize * sizeof(int32_t));
        int32_t preamble  = min((int32_t) (aligned - buffoff), filterout);
        int32_t rem_elems = filterout - preamble;

        if (laneid < preamble){
            buffoff[laneid] = wrapoutput[laneid];
        }

        for (int32_t k = laneid; k < rem_elems ; k += warpSize){
            aligned[k] = wrapoutput[preamble + k];
        }

        int32_t * cnts = buffer + ((4*warpSize-1)*(gridwidth+1));

        int32_t bnum0  = elems_old/(4*warpSize);
        int32_t bnum1  = (elems_old + filterout)/(4*warpSize);

        int32_t nset0  = (bnum0 == bnum1) ? filterout : (bnum1 * 4 * warpSize - elems_old);
        int32_t nset1  = filterout - nset0;

        int32_t totcnt0;
        if (laneid == 0) totcnt0 = atomicAdd(cnts + bnum0, nset0);
        totcnt0 = brdcst(totcnt0, 0) + nset0;

        int32_t totcnt1 = -1;
        if (nset1){
            if (laneid == 0) totcnt1 = atomicAdd(cnts + bnum1, nset1);
            totcnt1 = brdcst(totcnt1, 0) + nset1;
        }

        if (totcnt0 >= 4*warpSize){
            if (totcnt0 > 4*warpSize) printf("%d %d %d %d\n", elems_old, filterout, (gridwidth + 1) * (4 * warp_size - 1), totcnt0);
            assert(totcnt0 <= 4*warpSize);
            // output.push(buffer+bnum0*(4*warpSize));
            parent->consume_warp(buffer+bnum0*(4*warpSize), 4*warpSize);

            if (laneid == 0) cnts[bnum0] = 0; //clean up for next round
        }
        if (totcnt1 >= 4*warpSize){
            if (totcnt1 > 4*warpSize) printf("%d\n", totcnt1);
            assert(totcnt1 <= 4*warpSize);
            // output.push(buffer+bnum1*(4*warpSize));
            parent->consume_warp(buffer+bnum1*(4*warpSize), 4*warpSize);

            if (laneid == 0) cnts[bnum1] = 0; //clean up for next round
        }
    }
    if (warpid == 0) {
        int32_t * cnts = buffer + ((4*warpSize-1)*(gridwidth+1));

        int32_t finished_old;
        if (laneid == 0) finished_old = atomicAdd(&finished, 1);
        finished_old = brdcst(finished_old, 0);

        if (finished_old == gridwidth - 1){ //every other block has finished
            int32_t buffelems   = buffer_size;
            int32_t start       = round_down(buffelems, 4*warpSize);
            int32_t *buffstart  = buffer + start;

            // vec4 tmp_out;
            #pragma unroll
            for (int k = 0 ; k < 4 ; ++k) buffer[k*warpSize + laneid] = buffstart[k*warpSize + laneid];
            // reinterpret_cast<vec4*>(buffer)[laneid] = tmp_out;
            // parent->consume_warp(buffstart, buffelems - start);

            if (laneid == 0) {
                buffer_size                     = buffelems - start;
                finished                        = 0;
                cnts[buffelems/(4*warpSize)]    = 0;
                cnts[0]                         = buffelems - start;
                assert(buffelems - start <= 4*warpSize);
            }
        }
    }

    __syncthreads();

    parent->consume_close();
}

template<size_t warp_size, typename F, typename... T>
__device__ void unstable_select<warp_size, F, T...>::consume_open(){
    parent->consume_open();

    __syncthreads();

    extern __shared__ int32_t s[];

    const int32_t laneid     = get_laneid();
    const int32_t warpid     = get_warpid();
    // const int32_t width      = blockDim.x * blockDim.y;

    volatile int32_t *wrapoutbase   = ((int32_t *) s);
    volatile int32_t *wrapoutput    = wrapoutbase + 5 * warp_size * warpid;
    volatile int32_t *fcount        = wrapoutput  + 5 * warp_size - 1;

    if (laneid == 0) *fcount = 0;
}

template<size_t warp_size, typename F, typename... T>
__device__ void unstable_select<warp_size, F, T...>::at_open(){}

template<size_t warp_size, typename F, typename... T>
__device__ void unstable_select<warp_size, F, T...>::at_close(){
    if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0){
        const uint32_t laneid = get_laneid();

        parent->consume_open();

        __syncthreads();
        if (get_warpid() == 0){
            parent->consume_warp(buffer, buffer_size);
        }
        __syncthreads();

        parent->consume_close();
    }
}

template<size_t warp_size, typename F, typename... T>
__host__ void unstable_select<warp_size, F, T...>::before_open(){
    decltype(this->parent) p;
    gpu(hipMemcpy(&p, &(this->parent), sizeof(decltype(this->parent)), hipMemcpyDefault));
    p->open();
}

template<size_t warp_size, typename F, typename... T>
__host__ void unstable_select<warp_size, F, T...>::after_close(){
    decltype(this->parent) p;
    gpu(hipMemcpy(&p, &(this->parent), sizeof(decltype(this->parent)), hipMemcpyDefault));
    p->close();
}

template class unstable_select<WARPSIZE, less_eq_than<int32_t>, int32_t>;
