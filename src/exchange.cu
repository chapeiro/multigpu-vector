#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <algorithm>
#include "common.cuh"

using namespace std;

#ifndef NVAL
#define NVAL (10000*1024*4)
#endif

constexpr int N = NVAL;




__global__ void gen_test(hipEvent_t *store_to, hipEvent_t *ready, hipEvent_t *consumed){
    // gpu(hipEventCreate(store_to));
    gpu(hipEventRecord(*ready));
    gpu(hipEventSynchronize(*consumed));
}


__global__ void cons_test(hipEvent_t *store_to, hipEvent_t *ready, hipEvent_t *consumed){
    gpu(hipEventSynchronize(*ready));
    gpu(hipEventRecord(*consumed));
}


int32_t *a;
int32_t *b;

int main(){
    hipEvent_t store_to, ready, consumed;
    hipStream_t stra, strb;

    gpu(hipEventCreate(&ready));

    gpu(hipStreamCreate(&stra));
    gpu(hipStreamCreate(&strb));


    cons_test<<<1, 32, 0, stra>>>(&store_to, &ready, &consumed);
    gen_test <<<1, 32, 0, strb>>>(&store_to, &ready, &consumed);

    hipDeviceSynchronize();
    hipDeviceReset();

    return EXIT_SUCCESS;
}
