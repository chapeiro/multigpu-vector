
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdint>
#include <iostream>
#include <iomanip>
#include <chrono>
#include <cassert>
#include <cstring>
#include <algorithm>

using namespace std;

const int N = 10000*1024*4;

#define gpu(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

#define WARPSIZE (32)

#if __CUDA_ARCH__ < 300 || defined (NUSE_SHFL)
#define BRDCSTMEM(blockDim) ((blockDim.x * blockDim.y)/ WARPSIZE)
#else
#define BRDCSTMEM(blockDim) (0)
#endif

union vec4{
    int4 vec;
    int  i[4];
};

extern __shared__ int32_t s[];

template<typename T>
__device__ __host__ inline T round_up(T num, T mult){
    T rem = num % mult;
    if (rem == 0) return num;
    return num + mult - rem;
}



#if __CUDA_ARCH__ < 300 || defined (NUSE_SHFL)
template<typename T>
__device__ __forceinline__ T broadcast(T val, uint32_t src){
    uint32_t laneid;
    asm("mov.u32 %0, %%laneid;" : "=r"(laneid));

    volatile int32_t *bcount = (int32_t *) (s + 9 * blockDim.x * blockDim.y);
    uint32_t warpid;
    asm("mov.u32 %0, %%warpid;" : "=r"(warpid));

    if (laneid == src) bcount[warpid] = val;
    return bcount[warpid];
}
#else
   #define broadcast(v, l) (__shfl(v, l))
#endif


__device__ __forceinline__ void push_results(volatile int32_t *src, int32_t *dst, uint32_t* elems){
    uint32_t laneid;
    asm("mov.u32 %0, %%laneid;" : "=r"(laneid));

    uint32_t elems_old;
    if (laneid == 0) elems_old = atomicAdd(elems, 4*warpSize);
    elems_old = broadcast(elems_old, 0);

    vec4 tmp_out;
    #pragma unroll
    for (int k = 0 ; k < 4 ; ++k) tmp_out.i[k] = src[k*warpSize + laneid];
    reinterpret_cast<vec4*>(dst)[elems_old/4 + laneid] = tmp_out;
}

__global__ __launch_bounds__(65536, 4) void unstable_select(int32_t *src, int32_t *dst, int N, int32_t pred, int32_t *buffer, uint32_t *output_size, uint32_t *buffer_size){
    // int32_t *input  = (int32_t *) (s               );
    const int32_t width = blockDim.x * blockDim.y;
    const int32_t gridwidth = gridDim.x * gridDim.y;
    const int32_t bigwidth = width * gridwidth;
    volatile int32_t *output = (int32_t *) (s + 4*width);
    volatile int32_t *fcount = (int32_t *) (s + 9*width+BRDCSTMEM(blockDim));
    uint32_t *elems  = output_size;//(int32_t *) (s + 9*width+BRDCSTMEM(blockDim)+((blockDim.x * blockDim.y)/ WARPSIZE));

    const int32_t i       = threadIdx.x + threadIdx.y * blockDim.x;
    const int32_t blocki  = blockIdx.x  +  blockIdx.y *  gridDim.x;
    const int32_t laneid  = i % warpSize;

// #if __CUDA_ARCH__ < 300 || defined (NUSE_SHFL)
    const int32_t warpid  = i / warpSize;
// #endif
    const int32_t prevwrapmask = (1 << laneid) - 1;

    int32_t filterout = 0;

    volatile int32_t *wrapoutput = output + 5 * warpSize * warpid;

    //read from global memory
    for (int j = 0 ; j < N/4 ; j += bigwidth){
        bool predicate[4] = {false, false, false, false};
        vec4 tmp = reinterpret_cast<vec4*>(src)[i+j+blocki*width];

        #pragma unroll
        for (int k = 0 ; k < 4 ; ++k){
            if (4*(i+j+blocki*width)+k < N){
                // input[blockDim.x*k + i] = tmp.i[k];

                //compute predicate
                predicate[k] = tmp.i[k] <= pred;
            }
        }
        
        #pragma unroll
        for (int k = 0 ; k < 4 ; ++k){
            //aggreagate predicate results
            int32_t filter = __ballot(predicate[k]); //filter now contains set bits only for threads

            int32_t newpop = __popc(filter);

            //compute position of result
            if (predicate[k]){
                int32_t offset = filterout + __popc(filter & prevwrapmask);
                wrapoutput[offset] = tmp.i[k];//input[blockDim.x*k + i];
            }

            filterout += newpop;

            if (filterout >= 4*warpSize){
                push_results(wrapoutput, dst, elems);

                wrapoutput[laneid]             = wrapoutput[laneid + 4*warpSize];
                filterout                     -= 4*warpSize;
            }
        }
    }
    if (laneid == 0) fcount[warpid] = filterout;
    __syncthreads(); //this is needed to guarantee that all previous writes to dst are aligned

    for (int32_t m = 1 ; m <= 5 ; ++m){
        int32_t mask = (1 << m) - 1;
        if (!(warpid & mask)){
            int32_t target_wrapid               = warpid + (1 << (m - 1));
            int32_t target_filter_out           = fcount[target_wrapid];
            int32_t target_filter_out_rem       = target_filter_out;

            volatile int32_t *target_wrapoutput = output + 5 * warpSize * target_wrapid;

            for (int32_t k = 0; k < (target_filter_out + warpSize - 1)/warpSize ; ++k){
                if (laneid + k * warpSize < target_filter_out) {
                    wrapoutput[filterout + laneid] = target_wrapoutput[laneid + k * warpSize];
                }
                int32_t delta = min(target_filter_out_rem, warpSize);
                target_filter_out_rem -= delta;
                filterout += delta;

                if (filterout >= 4*warpSize){
                    push_results(wrapoutput, dst, elems);

                    wrapoutput[laneid]             = wrapoutput[laneid + 4*warpSize];
                    filterout                     -= 4*warpSize;
                }
            }
            fcount[warpid] = filterout;
        }
        __syncthreads();
    }
    if (warpid == 0 && filterout){
        int32_t elems_old;
        if (laneid == 0) elems_old = atomicAdd(buffer_size, filterout);
        elems_old = broadcast(elems_old, 0);

        volatile int32_t * buffoff = buffer  + elems_old;
        volatile int32_t * aligned = (int32_t *) round_up((uintptr_t) buffoff, warpSize * sizeof(int32_t));
        int32_t preamble  = min((int32_t) (aligned - buffoff), filterout);
        int32_t rem_elems = filterout - preamble;

        if (laneid < preamble){
            buffoff[laneid] = wrapoutput[laneid];
        }

        for (int32_t k = laneid; k < rem_elems ; k += warpSize){
            aligned[k] = wrapoutput[preamble + k];
        }

        int32_t * cnts = buffer + ((4*warpSize-1)*gridwidth);

        int32_t bnum0  = elems_old/(4*warpSize);
        int32_t bnum1  = (elems_old + filterout)/(4*warpSize);

        int32_t nset0  = (bnum0 == bnum1) ? filterout : (bnum1 * 4 * warpSize - elems_old);
        int32_t nset1  = filterout - nset0;

        int32_t totcnt0;
        if (laneid == 0) totcnt0 = atomicAdd(cnts + bnum0, nset0);
        totcnt0 = broadcast(totcnt0, 0) + nset0;

        int32_t totcnt1 = -1;
        if (nset1){
            if (laneid == 0) totcnt1 = atomicAdd(cnts + bnum1, nset1);
            totcnt1 = broadcast(totcnt1, 0) + nset1;
        }

        if (totcnt0 >= 4*warpSize) push_results(buffer+bnum0*(4*warpSize), dst, elems);
        if (totcnt1 >= 4*warpSize) push_results(buffer+bnum1*(4*warpSize), dst, elems);
    }
}

uint32_t unstable_select_gpu(int32_t *src, int32_t *dst, uint32_t N, int32_t pred, const dim3 &dimGrid, const dim3 &dimBlock, hipEvent_t &start, hipEvent_t &stop){
    int32_t *buffer;
    int32_t grid_size = dimGrid.x * dimGrid.y * dimGrid.z;

    gpu(hipMalloc((void**)&buffer, (grid_size * 4 * WARPSIZE + 2)* sizeof(int32_t)));

    uint32_t* counters = (uint32_t *) (buffer + grid_size * 4 * WARPSIZE);
    
    // initialize global counters
    gpu(hipMemset(buffer + grid_size * 4 * WARPSIZE - grid_size, 0, (grid_size + 2) * sizeof(int32_t)));

    hipEventRecord(start);

    size_t shared_mem = (9 * dimBlock.x * dimBlock.y + BRDCSTMEM(dimBlock) + ((dimBlock.x * dimBlock.y) / WARPSIZE))*sizeof(int32_t);

    // run kernel
    unstable_select<<<dimGrid, dimBlock, shared_mem>>>(src, dst, N, pred, buffer, counters, counters+1);
    
#ifndef NDEBUG
    gpu(hipPeekAtLastError()  );
    gpu(hipDeviceSynchronize());
#endif

    // wait to read counters from device
    uint32_t h_counters[2];
    gpu(hipMemcpy(h_counters, counters, 2 * sizeof(uint32_t), hipMemcpyDefault));
    uint32_t h_output_size = h_counters[0];
    uint32_t h_buffer_end  = h_counters[1];
    uint32_t h_buffer_start= (h_counters[1]/(4*WARPSIZE))*(4*WARPSIZE);
    uint32_t h_buffer_size = h_buffer_end - h_buffer_start;
    assert(h_buffer_start % (4*WARPSIZE) == 0);
    assert(h_buffer_end >= h_buffer_start);
    assert(h_buffer_size < 4*WARPSIZE);

    // combine results
    if (h_buffer_size > 0) hipMemcpy(dst+h_output_size, buffer+h_buffer_start, h_buffer_size * sizeof(int32_t), hipMemcpyDefault);
    hipEventRecord(stop);
    return h_output_size + h_buffer_size;
}
int32_t *a;
int32_t *b;

void stable_select_cpu(int32_t *a, int32_t *b, int N){
    int i = 0;
    for (int j = 0 ; j < N ; ++j) if (a[j] <= 50) b[i++] = a[j];
    b[i] = -1;
}

int main(){
    srand(time(0));

    a = (int32_t*) malloc(N*sizeof(int32_t));
    b = (int32_t*) malloc(N*sizeof(int32_t));

    for (int i = 0 ; i < N ; ++i) a[i] = rand() % 100 + 1;
    
    // char *ad;
    // int *bd;
    // const int csize = N*sizeof(char);
    // const int isize = N*sizeof(int);

    double millis = 0;
    {
        auto start = chrono::high_resolution_clock::now();
        stable_select_cpu(a, b, N);
        auto end   = chrono::high_resolution_clock::now();
        auto diff = end - start;

        millis = chrono::duration<double, milli>(diff).count();
        cout << millis << " ms" << endl;
    }
    int32_t *a_pinned;
    int32_t *b_pinned;
    hipEvent_t start, stop, start1, stop1, start2, stop2;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    dim3 dimBlock(1024, 1 );
    dim3 dimGrid( 8, 1 );

    hipHostMalloc((void**)&a_pinned, N*sizeof(int32_t), hipHostMallocDefault);
    hipHostMalloc((void**)&b_pinned, N*sizeof(int32_t), hipHostMallocDefault);

    memcpy(a_pinned, a, N*sizeof(int32_t));

#ifndef NTESTUVA
    int results1 = unstable_select_gpu(a_pinned, b_pinned, N, 50, dimGrid, dimBlock, start1, stop1);
#else
    int results1 = 0;
#endif

#ifndef NTESTMEMCPY
    int32_t *a_dev, *b_dev;

    gpu(hipMalloc( (void**)&a_dev, N*sizeof(int32_t)));
    gpu(hipMalloc( (void**)&b_dev, N*sizeof(int32_t)));

    hipEventRecord(start);
    gpu(hipMemcpy( a_dev, a_pinned, N*sizeof(int32_t), hipMemcpyDefault));
    
    int results2 = unstable_select_gpu(a_dev, b_dev, N, 50, dimGrid, dimBlock, start2, stop2);

    gpu(hipMemcpy(a_pinned, b_dev, N*sizeof(int32_t), hipMemcpyDefault));
    hipEventRecord(stop);

    gpu(hipFree(a_dev));
    gpu(hipFree(b_dev));
#else
    int results2 = 0;
#endif

    hipEventSynchronize(stop);

    hipDeviceSynchronize();
#ifndef NDEBUG
    int results = N;
    for (int i = 0 ; i < N ; ++i) {
        if (b[i] == -1) {
            results = i;
            break;
        } else {
            assert(b[i] <= 50);
            assert(b[i] > 0);
        }
    }
#ifndef NTESTUVA
    for (int i = 0 ; i < results1 ; ++i) {
        if (b_pinned[i] <= 0 || b_pinned[i] > 50){
            cout << b_pinned[i] << " " << i << endl;
        }
        // assert(b_pinned[i] <= 50);
        // assert(b_pinned[i] > 0);
    }
#endif
#ifndef NTESTMEMCPY
    for (int i = 0 ; i < results2 ; ++i) {
        if (a_pinned[i] <= 0 || a_pinned[i] > 50){
            cout << a_pinned[i] << " " << i << endl;
        }
        // assert(b_pinned[i] <= 50);
        // assert(b_pinned[i] > 0);
    }
#endif
    cout << results << " " << results1 << " " << results2 << " " << a_pinned[4] << endl;

    // assert(results1 == results2);
    if (results != results1){
        cout << "Wrong results!!!!!!" << endl;
    } else {
        sort(b_pinned, b_pinned + results);
        sort(b       , b        + results);
        for (int i = 0 ; i < results ; ++i){
            if (b[i] != b_pinned[i]){
                cout << "Wrong result: " << b_pinned[i] << " (vs " << b[i] << ") @" << i << " !!!!!!" << endl;
                exit(-1);
            }
        }
    }
#endif
    gpu(hipHostFree(a_pinned));
    gpu(hipHostFree(b_pinned));

    float milliseconds1 = 0;
    hipEventElapsedTime(&milliseconds1, start, stop);
    cout << milliseconds1 << endl;
    float milliseconds2 = 0;
    hipEventElapsedTime(&milliseconds2, start1, stop1);
    cout << milliseconds2 << endl;
    float milliseconds3 = 0;
    hipEventElapsedTime(&milliseconds3, start2, stop2);
    cout << milliseconds3 << endl;

    cout << endl;
    cout << millis/milliseconds1 << endl;
    cout << millis/milliseconds2 << endl;
    cout << millis/milliseconds3 << endl;
    
    hipDeviceSynchronize();
    hipDeviceReset();

    return EXIT_SUCCESS;
}
